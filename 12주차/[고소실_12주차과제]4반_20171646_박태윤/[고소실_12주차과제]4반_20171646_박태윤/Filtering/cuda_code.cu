#include "hip/hip_runtime.h"
#pragma once

#include "cuda_code.cuh"

#include <stdio.h>
#include<stdio.h>
#include<stdlib.h>
#include <math.h>
#include <Windows.h>
#include <time.h>
#include <assert.h>

#if USE_CPU_TIMER == 1
__int64 start, freq, end;
#define CHECK_TIME_START { QueryPerformanceFrequency((LARGE_INTEGER*)&freq); QueryPerformanceCounter((LARGE_INTEGER*)&start); }
#define CHECK_TIME_END(a) { QueryPerformanceCounter((LARGE_INTEGER*)&end); a = (float)((float)(end - start) / (freq / 1000.0f)); }
#else
#define CHECK_TIME_START
#define CHECK_TIME_END(a)
#endif

#if USE_GPU_TIMER == 1
hipEvent_t cuda_timer_start, cuda_timer_stop;
#define CUDA_STREAM_0 (0)

void create_device_timer()
{
	CUDA_CALL(hipEventCreate(&cuda_timer_start));
	CUDA_CALL(hipEventCreate(&cuda_timer_stop));
}

void destroy_device_timer()
{
	CUDA_CALL(hipEventDestroy(cuda_timer_start));
	CUDA_CALL(hipEventDestroy(cuda_timer_stop));
}

inline void start_device_timer()
{
	hipEventRecord(cuda_timer_start, CUDA_STREAM_0);
}

inline TIMER_T stop_device_timer()
{
	TIMER_T ms;
	hipEventRecord(cuda_timer_stop, CUDA_STREAM_0);
	hipEventSynchronize(cuda_timer_stop);

	hipEventElapsedTime(&ms, cuda_timer_start, cuda_timer_stop);
	return ms;
}

#define CHECK_TIME_INIT_GPU() { create_device_timer(); }
#define CHECK_TIME_START_GPU() { start_device_timer(); }
#define CHECK_TIME_END_GPU(a) { a = stop_device_timer(); }
#define CHECK_TIME_DEST_GPU() { destroy_device_timer(); }
#define BLOCK_SIZE 4

#else
#define CHECK_TIME_INIT_GPU()
#define CHECK_TIME_START_GPU()
#define CHECK_TIME_END_GPU(a)
#define CHECK_TIME_DEST_GPU()
#endif

TIMER_T compute_time = 0;
TIMER_T device_time = 0;

#define Window 2

__constant__ float constant_gaussian_kernel[25];

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//	
//	Gaussian ���͸��� �ϴ� Ŀ��
//	shared memory�� ������� �ʴ´�
//	
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void Gaussian_kernel_no_shared(IN unsigned char* d_bitmaps, OUT unsigned char* d_Gaussian, long width, long height) {

	const unsigned block_id = blockIdx.x;
	const unsigned thread_id = threadIdx.x;
	const unsigned r = block_id * BLOCK_SIZE + thread_id;

	int val;

	for (int c = 0; c < width; c++) {
		val = 0;
		for (int k = -2; k <= 2; k++) {
			for (int l = -2; l <= 2; l++) {
				if (c + l < 0 || c + l >= width || r + k < 0 || r + k >= height)
					continue;
				else 
					val += constant_gaussian_kernel[(k + 2) * 5 + l + 2] * d_bitmaps[(r + k) * width + c + l];
			}
		}

		d_Gaussian[c + width * r] = val;
	}
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//	
//	Gaussian ���͸��� �ϴ� Ŀ��
//	shared memory�� ����Ѵ�.
//	
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
extern __shared__ unsigned char sharedBuffer[];
__global__ void Gaussian_kernel_shared(INOUT unsigned char* d_bitmaps, OUT unsigned char* d_Gaussian, long width, long height) {
	/*Todo*/
	const unsigned block_id = blockIdx.x;
	const unsigned thread_id = threadIdx.x;
	const unsigned r = block_id * BLOCK_SIZE + thread_id;

	int idx = -2;

	if (thread_id == 0) {
		while (idx < 0) {
			for (int i = 0; i < width; i++) {
				if (r + idx < 0)
					sharedBuffer[(idx + 2) * width + i] = 0;
				else if (r + idx <= height)
					sharedBuffer[(idx + 2) * width + i] = d_bitmaps[(idx + r) * width + i];
			}
			idx++;
		}
	}

	idx = 1;

	if (thread_id == BLOCK_SIZE - 1) {
		while (idx < 3) {
			for (int i = 0; i < width; i++) {
				if (r + idx >= height)
					sharedBuffer[(thread_id + idx + 2) * width + i] = 0;
				else
					sharedBuffer[(thread_id + idx + 2) * width + i] = d_bitmaps[(r + idx) * width + i];
			}

			idx++;
		}
	}

	for (int i = 0; i < width; i++) 
		sharedBuffer[(thread_id + 2) * width + i] = d_bitmaps[r * width + i];

	__syncthreads();

	int val;

	for (int c = 0; c < width; c++) {
		val = 0;
		for (int k = -2; k <= 2; k++) {
			for (int l = -2; l <= 2; l++) {
				if (c + l < 0 || c + l >= width) 
					continue;
				else 
					val += sharedBuffer[(thread_id + 2 + k) * width + c + l] * constant_gaussian_kernel[(k + 2) * 5 + l + 2];
			}
		}

		d_Gaussian[c + width * r] = val;
	}


}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//	
//	Constant variable �� gaussian kernel�� �����ϴ� �Լ�
//	�Ŀ� gaussian filtering ���� ����Ѵ�.
//	
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

void Set_Gaussain_Kernel() {
	float _1 = 1.0f / 256.0f;
	float _4 = _1 * 4;
	float _6 = _1 * 6;
	float _16 = _1 * 16;
	float _24 = _1 * 24;
	float _36 = _1 * 36;

	float* p_gaussian_kernel = new float[25];

	p_gaussian_kernel[0] = p_gaussian_kernel[4] = p_gaussian_kernel[20] = p_gaussian_kernel[24] = _1;
	p_gaussian_kernel[1] = p_gaussian_kernel[3] = p_gaussian_kernel[5] = p_gaussian_kernel[9] = _4;
	p_gaussian_kernel[15] = p_gaussian_kernel[19] = p_gaussian_kernel[21] = p_gaussian_kernel[23] = _4;
	p_gaussian_kernel[2] = p_gaussian_kernel[10] = p_gaussian_kernel[14] = p_gaussian_kernel[22] = _6;
	p_gaussian_kernel[6] = p_gaussian_kernel[8] = p_gaussian_kernel[16] = p_gaussian_kernel[18] = _16;
	p_gaussian_kernel[7] = p_gaussian_kernel[11] = p_gaussian_kernel[13] = p_gaussian_kernel[17] = _24;
	p_gaussian_kernel[12] = _36;

	hipMemcpyToSymbol(HIP_SYMBOL(constant_gaussian_kernel), p_gaussian_kernel, sizeof(float) * 25);

	delete[] p_gaussian_kernel;
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//	
//	Ŀ���� �����ϱ� �� �ʿ��� �ڷ�� �غ� �� Ŀ���� ������ ����̽��� ����
//	Shared_flag �Է� �� NO_SHARED �� SHARED �� �� ���� ��ũ�θ� ������
//	flag���� �´� Ŀ���� ����
//	
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

float Do_Gaussian_on_GPU(IN unsigned char* p_bitmaps, OUT unsigned char* p_Gaussian, long width, long height, int Shared_flag)
{

	Set_Gaussain_Kernel();
	/*Todo*/
	hipError_t cudaStatus;

	CUDA_CALL(hipSetDevice(0));

	unsigned char* d_bitmaps, * d_Gaussian;
	size_t mem_size;

	mem_size = width * height * sizeof(unsigned char);
	CUDA_CALL(hipMalloc(&d_bitmaps, mem_size));
	CUDA_CALL(hipMalloc(&d_Gaussian, mem_size));

	CUDA_CALL(hipMemcpy(d_bitmaps, p_bitmaps, mem_size, hipMemcpyHostToDevice));

	dim3 blockDim(BLOCK_SIZE, 1);
	dim3 gridDim(height / blockDim.x, 1);
	CHECK_TIME_INIT_GPU();
	CHECK_TIME_START_GPU();

	switch (Shared_flag)
	{
	case NO_SHARED:
		Gaussian_kernel_no_shared << <gridDim, blockDim >> > (d_bitmaps, d_Gaussian, width, height);
		break;
	case SHARED:
		Gaussian_kernel_shared << <gridDim, blockDim, sizeof(unsigned char)* (BLOCK_SIZE + 2 * 2)* width >> > (d_bitmaps, d_Gaussian, width, height);
		break;
	}

	CUDA_CALL(cudaStatus = hipDeviceSynchronize());
	CHECK_TIME_END_GPU(device_time);
	CHECK_TIME_DEST_GPU();

	CUDA_CALL(hipMemcpy(p_Gaussian, d_Gaussian, mem_size, hipMemcpyDeviceToHost));

	hipFree(d_Gaussian);
	hipFree(d_bitmaps);

	return device_time;
}