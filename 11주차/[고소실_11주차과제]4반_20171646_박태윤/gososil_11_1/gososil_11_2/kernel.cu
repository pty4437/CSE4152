#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include<stdio.h>
#include<stdlib.h>
#include <math.h>
#include <Windows.h>
#include <time.h>
#include <assert.h>

#define CUDA_CALL(x) { const hipError_t a = (x); if(a != hipSuccess) { printf("\nCuda Error: %s (err_num=%d) at line:%d\n", hipGetErrorString(a), a, __LINE__); hipDeviceReset(); assert(0);}}
typedef float TIMER_T;

#define USE_CPU_TIMER 1
#define USE_GPU_TIMER 1

#if USE_CPU_TIMER == 1
__int64 start, freq, end;
#define CHECK_TIME_START { QueryPerformanceFrequency((LARGE_INTEGER*)&freq); QueryPerformanceCounter((LARGE_INTEGER*)&start); }
#define CHECK_TIME_END(a) { QueryPerformanceCounter((LARGE_INTEGER*)&end); a = (float)((float)(end - start) / (freq / 1000.0f)); }
#else
#define CHECK_TIME_START
#define CHECK_TIME_END(a)
#endif

#if USE_GPU_TIMER == 1
hipEvent_t cuda_timer_start, cuda_timer_stop;
#define CUDA_STREAM_0 (0)

void create_device_timer()
{
    CUDA_CALL(hipEventCreate(&cuda_timer_start));
    CUDA_CALL(hipEventCreate(&cuda_timer_stop));
}

void destroy_device_timer()
{
    CUDA_CALL(hipEventDestroy(cuda_timer_start));
    CUDA_CALL(hipEventDestroy(cuda_timer_stop));
}

inline void start_device_timer()
{
    hipEventRecord(cuda_timer_start, CUDA_STREAM_0);
}

inline TIMER_T stop_device_timer()
{
    TIMER_T ms;
    hipEventRecord(cuda_timer_stop, CUDA_STREAM_0);
    hipEventSynchronize(cuda_timer_stop);

    hipEventElapsedTime(&ms, cuda_timer_start, cuda_timer_stop);
    return ms;
}

#define CHECK_TIME_INIT_GPU() { create_device_timer(); }
#define CHECK_TIME_START_GPU() { start_device_timer(); }
#define CHECK_TIME_END_GPU(a) { a = stop_device_timer(); }
#define CHECK_TIME_DEST_GPU() { destroy_device_timer(); }
#else
#define CHECK_TIME_INIT_GPU()
#define CHECK_TIME_START_GPU()
#define CHECK_TIME_END_GPU(a)
#define CHECK_TIME_DEST_GPU()
#endif

#define BLOCK_SIZE 32
#define DATA_SIZE (1 << 20)

TIMER_T compute_time = 0;
TIMER_T device_time = 0;

int N;
float* A, * B, * C;
float* X0, * X1;
float* FX0, * FX1;
float* gpu_X0, * gpu_X1;
float* gpu_FX0, * gpu_FX1;


hipError_t find_roots_GPU(float* A, float* B, float* C, float* gpu_X0, float* gpu_X1, float* gpu_FX0, float* gpu_FX1);

__global__ void find_roots_Kernel(float* d_A, float* d_B, float* d_C, float* d_X0, float* d_X1, float* d_FX0, float* d_FX1)
{
    float a, b, c, d, x0, x1, tmp;
    int idx = threadIdx.x + blockDim.x * blockIdx.x;

    a = d_A[idx]; b = d_B[idx]; c = d_C[idx];
    d = sqrtf(b * b - 4.0f * a * c);
    tmp = 1.0f / (2.0f * a);
    d_X0[idx] = x0 = (-b - d) * tmp;
    d_X1[idx] = x1 = (-b + d) * tmp;
    d_FX0[idx] = (a * x0 + b) * x0 + c;
    d_FX1[idx] = (a * x1 + b) * x1 + c;

}


void find_roots_CPU(float* A, float* B, float* C, float* X0, float* X1, float* FX0, float* FX1, int n) {
    int i;
    float a, b, c, d, x0, x1, tmp;
    for (i = 0; i < n; i++) {
        a = A[i]; b = B[i]; c = C[i];
        d = sqrtf(b * b - 4.0f * a * c);
        tmp = 1.0f / (2.0f * a);
        X0[i] = x0 = (-b - d) * tmp;
        X1[i] = x1 = (-b + d) * tmp;
        FX0[i] = (a * x0 + b) * x0 + c;
        FX1[i] = (a * x1 + b) * x1 + c;
    }
}

void init_bin_file(int size) {
    srand((unsigned)time(NULL));
    printf("***Binary File init Start!!\n");
    FILE* fp_a = fopen("A.bin", "wb");
    FILE* fp_b = fopen("B.bin", "wb");
    FILE* fp_c = fopen("C.bin", "wb");


    fwrite(&size, sizeof(int), 1, fp_a);

    float random_num;
    for (int i = 0; i < size; i++) {
        random_num = (float)(rand() % 10) - 5.0f;
        if (random_num == 0.0f)
            random_num = 1.0f;
        fwrite(&random_num, sizeof(float), 1, fp_a);
    }

    fclose(fp_a);

    fwrite(&size, sizeof(int), 1, fp_b);
    random_num = 0.0f;

    for (int i = 0; i < size; i++) {
        random_num = (float)(rand() % 10) + 30.0f;
        if (random_num == 0.0f)
            random_num = 4.0f;
        fwrite(&random_num, sizeof(float), 1, fp_b);
    }

    fclose(fp_b);

    fwrite(&size, sizeof(int), 1, fp_c);

    for (int i = 0; i < size; i++) {
        random_num = (float)(rand() % 10) - 5.0f;
        if (random_num == 0.0f)
            random_num = 2.0f;
        fwrite(&random_num, sizeof(float), 1, fp_c);
    }

    fclose(fp_c);


    printf("***Binary File init End!!\n\n");
}

void read_bin_file()
{
    printf("***Binary File Read Start!!\n");
    FILE* fp = fopen("A.bin", "rb");
    /*Todo*/

    fread(&N, sizeof(int), 1, fp);

    A = new float[N]; B = new float[N]; C = new float[N];
    X0 = new float[N]; X1 = new float[N];
    FX0 = new float[N]; FX1 = new float[N];
    gpu_X0 = new float[N]; gpu_X1 = new float[N];
    gpu_FX0 = new float[N]; gpu_FX1 = new float[N];

    fread(A, sizeof(float), N, fp);
    

    fclose(fp);

    fp = fopen("B.bin", "rb");
    fread(&N, sizeof(int), 1, fp);
    fread(B, sizeof(float), N, fp);
    fclose(fp);

    fp = fopen("C.bin", "rb");
    fread(&N, sizeof(int), 1, fp);
    fread(C, sizeof(float), N, fp);
    fclose(fp);

    printf("***Binary File Read End!!\n\n");
}

int main()
{
    int i;
    FILE* fp;

    init_bin_file(DATA_SIZE);
    read_bin_file();

    printf("==========================================\n");
    printf("N is %d.\n", N);
    printf("==========================================\n\n");

    printf("==========================================\n");
    printf("***find_roots_CPU Start!!\n");
    CHECK_TIME_START;
    find_roots_CPU(A, B, C, X0, X1, FX0, FX1, N);
    CHECK_TIME_END(compute_time);
    printf("***find_roots_CPU End!!\n");
    printf("CPU time = %.6f\n", compute_time);
    //printf("X0 : %f, FX0 : %f\n", X0[1], FX0[1]);
    //printf("X1 : %f, FX1 : %f\n", X1[1], FX1[1]);
    printf("==========================================\n\n");


    printf("==========================================\n");
    printf("***find_roots_GPU Start!!\n");
    find_roots_GPU(A, B, C, gpu_X0, gpu_X1, gpu_FX0, gpu_FX1);
    printf("***find_roots_GPU End!!\n");
    printf("GPU time = %.6f\n", device_time);
    //printf("X0 : %f, FX0 : %f\n", gpu_X0[1], gpu_FX0[1]);
    //printf("X1 : %f, FX1 : %f\n", gpu_X1[1], gpu_FX1[1]);
    printf("==========================================\n\n");

    fp = fopen("X0.bin", "wb");
    if (!fp) {
        fprintf(stderr, "Error: cannot open the output file...\n");
        exit(-1);
    }

    fwrite(&N, sizeof(int), 1, fp);

    for (i = 0; i < N; i++)
        fwrite(&gpu_X0[i], sizeof(float), 1, fp);

    fclose(fp);


    fp = fopen("X1.bin", "wb");
    if (!fp) {
        fprintf(stderr, "Error: cannot open the output file...\n");
        exit(-1);
    }

    fwrite(&N, sizeof(int), 1, fp);

    for (i = 0; i < N; i++)
        fwrite(&gpu_X1[i], sizeof(float), 1, fp);

    fclose(fp);

    fp = fopen("FX0.bin", "wb");

    fwrite(&N, sizeof(int), 1, fp);

    for (i = 0; i < N; i++)
        fwrite(&gpu_FX0[i], sizeof(float), 1, fp);

    fclose(fp);

    printf("end!!\n\n");

    fp = fopen("FX1.bin", "wb");

    fwrite(&N, sizeof(int), 1, fp);

    for (i = 0; i < N; i++)
        fwrite(&gpu_X0[i], sizeof(float), 1, fp);

    fclose(fp);


    return 0;
}


hipError_t find_roots_GPU(float* A, float* B, float* C, float* gpu_X0, float* gpu_X1, float* gpu_FX0, float* gpu_FX1)
{
    CHECK_TIME_INIT_GPU();

    hipError_t cudaStatus;
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    float* d_A, * d_B, * d_C;
    float* d_X0, * d_X1;
    float* d_FX0, * d_FX1;
    dim3 dimBlock(BLOCK_SIZE, 1);
    dim3 dimGrid(N / dimBlock.x, 1);

    size_t size;

    size = N * sizeof(float);

    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);
    hipMalloc(&d_X0, size);
    hipMalloc(&d_X1, size);
    hipMalloc(&d_FX0, size);
    hipMalloc(&d_FX1, size);

    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);
    hipMemcpy(d_C, C, size, hipMemcpyHostToDevice);   

    size = N;

    CHECK_TIME_START_GPU();

    find_roots_Kernel << < dimGrid, dimBlock >> > (d_A, d_B, d_C, d_X0, d_X1, d_FX0, d_FX1);

    CHECK_TIME_END_GPU(device_time);
    size = N * sizeof(float);

    hipGetLastError();
    hipDeviceSynchronize();

    hipMemcpy(gpu_X0, d_X0, size, hipMemcpyDeviceToHost);
    hipMemcpy(gpu_X1, d_X1, size, hipMemcpyDeviceToHost);
    hipMemcpy(gpu_FX0, d_FX0, size, hipMemcpyDeviceToHost);
    hipMemcpy(gpu_FX1, d_FX1, size, hipMemcpyDeviceToHost);

    CHECK_TIME_DEST_GPU();

Error:
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipFree(d_X0);
    hipFree(d_X1);
    hipFree(d_FX0);
    hipFree(d_FX1);

    return cudaStatus;
}